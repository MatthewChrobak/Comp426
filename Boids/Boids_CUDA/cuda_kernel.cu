#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "DataTypes.h"

__global__ void updateBoid(Boid* boids)
{
	int boidId = blockIdx.x * blockDim.x + threadIdx.x;
	auto boid = boids + boidId;

	boid->Velocity.X = boid->NewVelocity.X;
	boid->Velocity.Y = boid->NewVelocity.Y;

	boid->Position.X += boid->Velocity.X;
	boid->Position.Y += boid->Velocity.Y;
}

void updateFlocks(Boid* boids, int numFlocks, int numBoids)
{
	Boid* MBoids;
	hipMalloc((void**)&MBoids, sizeof(Boid) * numFlocks * numBoids);
	hipMemcpy(MBoids, boids, numBoids * numFlocks * sizeof(Boid), hipMemcpyHostToDevice);
	updateBoid<<<numFlocks, numBoids>>>(MBoids);
	hipDeviceSynchronize();
	hipMemcpy(boids, MBoids, numBoids * numFlocks * sizeof(Boid), hipMemcpyDeviceToHost);
	hipFree(MBoids);
}