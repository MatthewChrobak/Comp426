#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "Boid.h"
#include <iostream>


__global__ void updateBoid(Boid** boids)
{
	int boidId = blockIdx.x * blockDim.x + threadIdx.x;
	Boid* boid = *(boids + boidId);
	
	int newpos = boid->Position.X + boid->Velocity.X;
	printf("Boids are at: %f\n", newpos);
	//boid->Position.Y += boid->Velocity.Y;
}

void updateFlocks(Boid** boids, int numFlocks, int numBoids)
{
	updateBoid<<<numFlocks, numBoids>>>(boids);
	hipDeviceSynchronize();
}