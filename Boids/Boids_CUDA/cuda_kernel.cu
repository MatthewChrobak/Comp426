#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "DataTypes.h"

__global__ void updateBoid(Boid* boids, int numFlocks, int numBoids)
{
	int boidId = blockIdx.x * blockDim.x + threadIdx.x;
	auto boid = boids + boidId;
	
	boid->Position.X += boid->Velocity.X;
	boid->Position.Y += boid->Velocity.Y;
}

void updateFlocks(Boid* boids, int numFlocks, int numBoids)
{
	Boid* MBoids;
	hipMalloc((void**)&MBoids, sizeof(Boid) * numFlocks * numBoids);
	Boid* localBoids = (Boid*)malloc(numFlocks * numBoids * sizeof(Boid));

	hipMemcpy(MBoids, boids, numBoids * numFlocks * sizeof(Boid), hipMemcpyHostToDevice);
	updateBoid<<<numFlocks, numBoids >>>(MBoids, numFlocks, numBoids);
	hipDeviceSynchronize();
	hipMemcpy(localBoids, MBoids, numBoids * numFlocks * sizeof(Boid), hipMemcpyDeviceToHost);

	for (int i = 0; i < numBoids * numFlocks; i++) {
		*(boids + i) = *(localBoids + i);
	}
	free(localBoids);
	hipFree(MBoids);
}